#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
 
#define N 1000000
 
__global__ void add(int *a, int *b, int*c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}
 
int main(void)
{
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
 
	// CPU側でメモリを割り当てる
	a = (int *)malloc(N * sizeof(int));
	b = (int *)malloc(N * sizeof(int));
	c = (int *)malloc(N * sizeof(int));
 
	// GPU側でメモリを割り当てる
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
 
	// CPU側で配列aと配列bを設定する
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = 2 * i;
	}
 
	// 配列aと配列bをGPUにコピーする
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
 
	add << <128, 128 >> >(dev_a, dev_b, dev_c);
 
	// 配列cをGPUからCPUにコピーする
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
 
	// 要求した処理をGPUが行ったことを確認する
	bool success = true;
	for (int i = 0; i < N; i++)
	{
		if ((a[i] + b[i]) != c[i])
		{
			printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
	}
	if (success)
	{
		printf("We did it !\n");
	}
 
	// GPU側で割り当てたメモリを開放する
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
 
	// CPU側で割り当てたメモリを解放する
	free(a);
	free(b);
	free(c);
 
	int i;
	scanf("%d", &i);
 
	return 0;
}